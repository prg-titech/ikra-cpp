#include "hip/hip_runtime.h"
// N-Body Simulation
// Code adapted from: http://physics.princeton.edu/~fpretori/Nbody/code.htm

#include <cstdio>
#include <cstdlib>
#include <cmath>

#include "benchmark.h"
#include "executor/cuda_executor.h"

using namespace std;
using IdType = uintptr_t;

static const int kIterations = 5;
static const int kNumBodies = 8000;
static const double kMaxMass = 1000;
static const double kTimeInterval = 0.5;

static const double kGravityConstant = 6.673e-11;   // gravitational constant

struct Container {
  double a_Body_mass[kNumBodies];
  double a_Body_position_0[kNumBodies];
  double a_Body_position_1[kNumBodies];
  double a_Body_velocity_0[kNumBodies];
  double a_Body_velocity_1[kNumBodies];
  double a_Body_force_0[kNumBodies];
  double a_Body_force_1[kNumBodies];
};

__device__ Container d_container;
Container h_container;

#define RAND (1.0 * rand() / RAND_MAX)

void Body_initialize(IdType i, double mass, double pos_x, double pos_y,
                     double vel_x, double vel_y) {
  h_container.a_Body_mass[i] = mass;
  h_container.a_Body_position_0[i] = pos_x;
  h_container.a_Body_position_1[i] = pos_y;
  h_container.a_Body_velocity_0[i] = vel_x;
  h_container.a_Body_velocity_1[i] = vel_y;
  h_container.a_Body_force_0[i] = 0.0;
  h_container.a_Body_force_1[i] = 0.0;
}

__device__ void Body_add_force(IdType self, IdType body) {
  if (self == body) return;
  double EPS = 0.01;    // Softening parameter (just to avoid infinities).
  double dx = d_container.a_Body_position_0[body]
              - d_container.a_Body_position_0[self];
  double dy = d_container.a_Body_position_1[body]
              - d_container.a_Body_position_1[self];
  double dist = sqrt(dx*dx + dy*dy);
  double F = kGravityConstant
             * d_container.a_Body_mass[self]*d_container.a_Body_mass[body]
             / (dist*dist + EPS*EPS);
  d_container.a_Body_force_0[self] += F*dx / dist;
  d_container.a_Body_force_1[self] += F*dy / dist;
}

__device__  void Body_add_all_forces_to_this(IdType self) {
  for (IdType i = 0; i < kNumBodies; ++i) {
    Body_add_force(self, i);
  }
}

__global__ void kernel_Body_add_all_forces_to_this() {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < kNumBodies) {
    Body_add_all_forces_to_this(tid);
  }
}

__device__ void Body_reset_force(IdType self) {
  d_container.a_Body_force_0[self] = 0.0;
  d_container.a_Body_force_1[self] = 0.0;
}

__global__ void kernel_Body_reset_force() {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < kNumBodies) {
    Body_reset_force(tid);
  }
}

__device__ void Body_update(IdType self, double dt) {
  d_container.a_Body_velocity_0[self] +=
      d_container.a_Body_force_0[self]*dt / d_container.a_Body_mass[self];
  d_container.a_Body_velocity_1[self] +=
      d_container.a_Body_force_1[self]*dt / d_container.a_Body_mass[self];
  d_container.a_Body_position_0[self] +=
      d_container.a_Body_velocity_0[self]*dt;
  d_container.a_Body_position_1[self] +=
      d_container.a_Body_velocity_1[self]*dt;

  if (d_container.a_Body_position_0[self] < -1
        || d_container.a_Body_position_0[self] > 1) {
    d_container.a_Body_velocity_0[self] = -d_container.a_Body_velocity_0[self];
  }
  if (d_container.a_Body_position_1[self] < -1
        || d_container.a_Body_position_1[self] > 1) {
    d_container.a_Body_velocity_1[self] = -d_container.a_Body_velocity_1[self];
  }
}

__global__ void kernel_Body_update(double dt) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < kNumBodies) {
    Body_update(tid, dt);
  }
}

__device__ void Body_codegen_simple_update(IdType self, double dt) {
  for (int i = 0; i < 1000; ++i) {
    d_container.a_Body_velocity_0[self] +=
        d_container.a_Body_force_0[self]*dt / d_container.a_Body_mass[self];
    d_container.a_Body_velocity_1[self] +=
        d_container.a_Body_force_1[self]*dt / d_container.a_Body_mass[self];
    d_container.a_Body_position_0[self] +=
        d_container.a_Body_velocity_0[self]*dt;
    d_container.a_Body_position_1[self] +=
        d_container.a_Body_velocity_1[self]*dt;
  }
}

__global__ void kernel_Body_codegen_simple_update(double dt) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < kNumBodies) {
    Body_codegen_simple_update(tid, dt);
  }
}

void instantiation() {
  srand(42);

  // Create objects.
  for (IdType i = 0; i < kNumBodies; ++i) {
    double mass = (RAND/2 + 0.5) * kMaxMass;
    double pos_x = RAND*2 - 1;
    double pos_y = RAND*2 - 1;
    double vel_x = (RAND - 0.5) / 1000;
    double vel_y = (RAND - 0.5) / 1000;
    Body_initialize(i, mass, pos_x, pos_y, vel_x, vel_y);
  }

  // Transfer data to GPU.
  hipMemcpyToSymbol(HIP_SYMBOL(d_container), &h_container, sizeof(Container), 0,
                     hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  assert(hipPeekAtLastError() == hipSuccess);
}

void run_simulation() {
  uintptr_t num_blocks = ikra::executor::cuda::cuda_blocks_1d(kNumBodies);
  uintptr_t num_threads = ikra::executor::cuda::cuda_threads_1d(kNumBodies);

  for (int i = 0; i < kIterations; ++i) {
    // Reset forces.
    kernel_Body_reset_force<<<num_blocks, num_threads>>>();

    // Update forces.
    kernel_Body_add_all_forces_to_this<<<num_blocks, num_threads>>>();

    // Update velocities and positions.
    kernel_Body_update<<<num_blocks, num_threads>>>(kTimeInterval);
  }

  hipDeviceSynchronize();
  assert(hipPeekAtLastError() == hipSuccess);
}

void run_simple() {
  uintptr_t num_blocks = ikra::executor::cuda::cuda_blocks_1d(kNumBodies);
  uintptr_t num_threads = ikra::executor::cuda::cuda_threads_1d(kNumBodies);

  for (int i = 0; i < kIterations*100; ++i) {
    kernel_Body_codegen_simple_update<<<num_blocks, num_threads>>>(
        kTimeInterval);
  }

  hipDeviceSynchronize();
  assert(hipPeekAtLastError() == hipSuccess);
}

int main() {
  uint64_t time_instantiation = measure<>::execution(instantiation);
  gpuErrchk(hipPeekAtLastError());

  uint64_t time_simulation = measure<>::execution(run_simulation);
  gpuErrchk(hipPeekAtLastError());

  uint64_t time_simple = measure<>::execution(run_simple);
  gpuErrchk(hipPeekAtLastError());

  // Transfer data to CPU.
  hipMemcpyFromSymbol(&h_container, HIP_SYMBOL(d_container), sizeof(Container), 0,
                       hipMemcpyDeviceToHost);
  gpuErrchk(hipPeekAtLastError());

  // Calculate checksum
  int checksum = 11;
  for (uintptr_t i = 0; i < kNumBodies; i++) {
    checksum += reinterpret_cast<int>(
        r_float2int(h_container.a_Body_position_0[i]));
    checksum += reinterpret_cast<int>(
        r_float2int(h_container.a_Body_position_1[i]));
    checksum = checksum % 1234567;

    if (i < 10) {
      printf("VALUE[%lu] = %f, %f\n", i,
             h_container.a_Body_position_0[i],
             h_container.a_Body_position_1[i]);
    }
  }

  printf("instantiation: %lu\nsimulation: %lu\nsimple: %lu\nchecksum: %i\n",
         time_instantiation, time_simulation, time_simple, checksum);
  return 0;
}

__global__  void codengen_simple_update(IdType body, double dt) {
  Body_codegen_simple_update(body, dt);
}
