#define NDEBUG

#define NUM_INST 524288
#define ARRAY_SIZE 48

#include "executor/cuda_executor.h"
#include "soa/soa.h"

#include "benchmark.h"

using ikra::soa::IndexType;
using ikra::soa::SoaLayout;
using ikra::executor::cuda::construct;

#define CUDA_THREAD_ID (threadIdx.x + blockIdx.x * blockDim.x)

class DummyClass : public SoaLayout<DummyClass, NUM_INST> {
 public:
  IKRA_INITIALIZE_CLASS

  __device__ DummyClass(int f0, int f2): field0(f0), field2(f2) {
    for (int i = 0; i < ARRAY_SIZE; ++i) {
      field1[i] = CUDA_THREAD_ID*17 + i;
    }
  }

  int_ field0;

  // Array has size 12 bytes.
  array_(int, ARRAY_SIZE, soa) field1;

  int_ field2;

  __device__ void update_field1(int increment) {
    for (int i = 0; i < ARRAY_SIZE; ++i) {
      field1[i] += increment + field0 + field2;
    }
  }
};

IKRA_DEVICE_STORAGE(DummyClass);


void action() {
  DummyClass::initialize_storage();
  DummyClass* first = construct<DummyClass>(NUM_INST, 29, 1357);
  gpuErrchk(hipPeekAtLastError());

  cuda_execute(&DummyClass::update_field1, first, NUM_INST, 19);
  gpuErrchk(hipPeekAtLastError());

  hipDeviceSynchronize();
}

void run_test_construct_and_execute() {
  uint64_t time_action = measure<>::execution(action);
  printf("Time for action: %lu\n", time_action);

#ifndef NDEBUG
  // Check result (some samples).
  for (int k = 0; k < 100; ++k) {
    int i = rand() % NUM_INST;
    for (int j = 0; j < ARRAY_SIZE; ++j) {
      int actual1 = DummyClass::get(i)->field1[j];
      int expected1 = i*17 + j + 19 + 29 + 1357;
      if (actual1 != expected1) {
        printf("Wrong result! Expected %i, but found %i\n",
               expected1, actual1);
        exit(1);
      }
    }
  }
#endif  // NDEBUG
}

int main() {
  run_test_construct_and_execute();
}
