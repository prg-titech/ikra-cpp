#include <array>

#include "gtest/gtest.h"

#include "executor/cuda_executor.h"
#include "soa/soa.h"

using ikra::soa::IndexType;
using ikra::soa::SoaLayout;
using ikra::executor::cuda::construct;
using ikra::executor::cuda::KernelConfig;
using ikra::executor::cuda::KernelConfiguration;

const static int kTestSize = 12;

class DummyClass : public SoaLayout<DummyClass, 1000> {
 public:
  IKRA_INITIALIZE_CLASS

  __host__ __device__ DummyClass(int f0, int f1) : field0(f0), field1(f1) {}

  int_ field0;
  int_ field1;

  template<int W_SZ>
  __device__ void add_fields(int increment) {
    field0 = field0 + field1 + increment + this->id() + W_SZ;
  }
};

IKRA_DEVICE_STORAGE(DummyClass);


// Cannot run "cuda_execute" inside gtest case.
void run_test_outer_cuda_execute_configuration() {
  DummyClass::initialize_storage();
  EXPECT_EQ(DummyClass::size(), 0UL);

  DummyClass* first = construct<DummyClass>(kTestSize, 5, 6);
  gpuErrchk(hipPeekAtLastError());

  // Use a virtual warp size of 4.
  cuda_execute_vw(&DummyClass::add_fields, KernelConfiguration<4>(12),
                  first, 12, 10);

  // Check result.
  for (int i = 0; i < kTestSize; ++i) {
    int actual = DummyClass::get(i)->field0;
    int expected = 10 + 5 + 6 + i + 4;
    EXPECT_EQ(actual, expected);
  }

  // Copy size to host memory and compare.
  EXPECT_EQ(DummyClass::size(), static_cast<IndexType>(kTestSize));

  // Make sure that we had no CUDA failures.
  gpuErrchk(hipPeekAtLastError());
}

void run_test_outer_cuda_execute_strategy() {
  DummyClass::initialize_storage();
  EXPECT_EQ(DummyClass::size(), 0UL);

  DummyClass* first = construct<DummyClass>(kTestSize, 5, 6);
  gpuErrchk(hipPeekAtLastError());

  cuda_execute_vw(&DummyClass::add_fields, KernelConfig<>::standard(),
                  first, 12, 10);

  // Check result.
  for (int i = 0; i < kTestSize; ++i) {
    int actual = DummyClass::get(i)->field0;
    int expected = 10 + 5 + 6 + i + 1;
    EXPECT_EQ(actual, expected);
  }

  // Copy size to host memory and compare.
  EXPECT_EQ(DummyClass::size(), static_cast<IndexType>(kTestSize));

  // Make sure that we had no CUDA failures.
  gpuErrchk(hipPeekAtLastError());
}

void run_test_outer_cuda_execute_none() {
  DummyClass::initialize_storage();
  EXPECT_EQ(DummyClass::size(), 0UL);

  DummyClass* first = construct<DummyClass>(kTestSize, 5, 6);
  gpuErrchk(hipPeekAtLastError());

  cuda_execute_vw(&DummyClass::add_fields,
                  first, 12, 10);

  // Check result.
  for (int i = 0; i < kTestSize; ++i) {
    int actual = DummyClass::get(i)->field0;
    int expected = 10 + 5 + 6 + i + 1;
    EXPECT_EQ(actual, expected);
  }

  // Copy size to host memory and compare.
  EXPECT_EQ(DummyClass::size(), static_cast<IndexType>(kTestSize));

  // Make sure that we had no CUDA failures.
  gpuErrchk(hipPeekAtLastError());
}

TEST(VirtualWarpTest, TestOuterCudaExecuteStrategy) {
  run_test_outer_cuda_execute_strategy();
}

TEST(VirtualWarpTest, TestOuterCudaExecuteNone) {
  run_test_outer_cuda_execute_none();
}

TEST(VirtualWarpTest, TestOuterCudaExecuteConfiguration) {
  run_test_outer_cuda_execute_configuration();
}
