#include "hip/hip_runtime.h"
// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010
 
#include <stdio.h>
//#include "executor/cuda_executor.h"
#include "soa/soa.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using ikra::soa::IndexType;
using ikra::soa::SoaLayout;
using ikra::soa::kAddressModeZero;
using ikra::soa::DynamicStorage;

__device__ char data_buffer[10000];



class Vertex : public SoaLayout<Vertex, 1000> {
 public:
  IKRA_INITIALIZE_CLASS(data_buffer)

  __device__ Vertex(int a) {
    printf("IN CONSTRUCTOR!!\n");
  }


/*
  int_ field0;
  int_ field1;

  __ikra_device__ void add_fields() {
    field0 = field0 + field1;
  }
*/

};


template<typename T>
__global__ void myKernel() {
  new (Vertex::get_(0)) Vertex(123);
}



int main()
{
  void* bla;
  hipMalloc( (void**)&bla, 100 );
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
  
  myKernel<int><<<1,2>>>();
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  printf("!!!!!\n");
}


// Keep nullptr as special "not an object"