#include "hip/hip_runtime.h"
// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010
 
#include <stdio.h>
#include "executor/executor.h"
#include "soa/soa.h"

const int N = 16; 
const int blocksize = 16; 

using ikra::soa::IndexType;
using ikra::soa::SoaLayout;
using ikra::soa::kAddressModeZero;
using ikra::soa::DynamicStorage;
using ikra::executor::execute;
using ikra::executor::execute_and_reduce;

char data_buffer[10000];

class Vertex : public SoaLayout<Vertex, 1000> {
 public:
  IKRA_INITIALIZE_CLASS(data_buffer)

  int_ field0;
  int_ field1;
};

__global__ 
void hello(char *a, int *b) 
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  Vertex::get(tid)->field0 += Vertex::get(tid)->field1;
}
 
int main()
{
  char a[N] = "Hello \0\0\0\0\0\0";
  int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
  char *ad;
  int *bd;
  const int csize = N*sizeof(char);
  const int isize = N*sizeof(int);
 
  printf("%s", a);
 
  hipMalloc( (void**)&ad, csize ); 
  hipMalloc( (void**)&bd, isize ); 
  hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
  hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
  
  dim3 dimBlock( blocksize, 1 );
  dim3 dimGrid( 1, 1 );
  hello<<<dimGrid, dimBlock>>>(ad, bd);
  hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
  hipFree( ad );
  hipFree( bd );
  
  printf("%s\n", a);
  return EXIT_SUCCESS;
}
