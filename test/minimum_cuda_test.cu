#include "gtest/gtest.h"

#include "executor/cuda_executor.h"
#include "soa/soa.h"

using ikra::soa::SoaLayout;
using ikra::executor::cuda::construct;

const static int kTestSize = 12;

__device__ char data_buffer[10000];


class Vertex : public SoaLayout<Vertex, 1000> {
 public:
  IKRA_INITIALIZE_CLASS(data_buffer)

  __device__ Vertex(int f0, int f1) : field0(f0), field1(f1) {}

  int_ field0;
  int_ field1;

  __device__ void add_fields(int increment) {
    field0 = field0 + field1 + increment + this->id();

    Vertex::get(0)->field0.get();
  }
};

// Cannot run "cuda_execute" inside gtest case.
void run_test() {
  Vertex::cuda_initialize_storage();

  Vertex* first = construct<Vertex>(kTestSize, 5, 6);
  cuda_execute(Vertex, add_fields, kTestSize, first, 10)

  // Check result.
  for (int i = 0; i < kTestSize; ++i) {
    int actual = Vertex::get_uninitialized(i)->field0;
    int expected = 10 + 5 + 6 + i;
    EXPECT_EQ(actual, expected);
  }

  // Make sure that we had no CUDA failures.
  gpuErrchk(hipPeekAtLastError());
}

TEST(MinimumCudaTest, ConstructAndExecute) {
  run_test();
}
