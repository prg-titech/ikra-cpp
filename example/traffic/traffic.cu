#include "hip/hip_runtime.h"
#include "executor/executor.h"
#include "soa/soa.h"
#include "executor/cuda_executor.h"

static const uint32_t kNumCells = 217000;
static const uint32_t kNumCars = 21000;

static const uint32_t kArrayInlineSizeOutgoingCells = 4;
static const uint32_t kArrayInlineSizeIncomingCells = 4;
static const uint32_t kArrayInlineSizePath = 6;

using ikra::soa::SoaLayout;
using ikra::soa::kAddressModeZero;
using ikra::soa::StaticStorageWithArena;

class Car;

class Cell : public SoaLayout<Cell, kNumCells> {
 public:
  IKRA_INITIALIZE_CLASS

  enum Type {
    // Sorted from smallest to largest.
    kResidential,
    kTertiary,
    kSecondary,
    kPrimary,
    kMotorwayLink,
    kMotorway,

    kMaxType
  };

  __host__ __device__ Cell(uint32_t max_velocity, double x, double y,
                           uint32_t num_incoming, Cell** incoming,
                           uint32_t num_outgoing, Cell** outgoing,
                           Car* car, bool is_free, bool is_sink,
                           Type type = kResidential)
      : max_velocity_(max_velocity), x_(x), y_(y), type_(type),
        num_incoming_cells_(num_incoming), num_outgoing_cells_(num_outgoing),
        incoming_cells_(num_incoming), outgoing_cells_(num_outgoing),
        car_(car), is_free_(is_free), is_sink_(is_sink) {
    for (uint32_t i = 0; i < num_incoming; ++i) {
      incoming_cells_[i] = incoming[i];
    }

    for (uint32_t i = 0; i < num_outgoing; ++i) {
      outgoing_cells_[i] = outgoing[i];
    }

    controller_max_velocity_ = max_velocity_;
  }

  // Overload: Provide cell indices instead of pointers.
  __host__ __device__ Cell(uint32_t max_velocity, double x, double y,
                           uint32_t num_incoming, unsigned int* incoming,
                           uint32_t num_outgoing, unsigned int* outgoing,
                           Car* car, bool is_free, bool is_sink,
                           Type type = kResidential)
      : max_velocity_(max_velocity), x_(x), y_(y), type_(type),
        num_incoming_cells_(num_incoming), num_outgoing_cells_(num_outgoing),
        incoming_cells_(num_incoming), outgoing_cells_(num_outgoing),
        car_(car), is_free_(is_free), is_sink_(is_sink) {
    for (uint32_t i = 0; i < num_incoming; ++i) {
      incoming_cells_[i] = Cell::get_uninitialized(incoming[i]);
    }

    for (uint32_t i = 0; i < num_outgoing; ++i) {
      outgoing_cells_[i] = Cell::get_uninitialized(outgoing[i]);
    }

    controller_max_velocity_ = max_velocity_;
  }

  // A cell is free if is does not contain a car.
  bool_ is_free_;
  __device__ bool is_free() const { return is_free_; }

  // A cell is usually a sink if does not have any outgoing edges.
  bool_ is_sink_;
  __device__ bool is_sink() const { return is_sink_; }

  // Return the maximum velocity that is allowed on this street in general.
  uint32_t_ max_velocity_;

  // Return max. velocity allowed with respect to traffic controllers.
  uint32_t_ controller_max_velocity_;

  // Returns the maximum velocity allowed on this cell at this moment. This
  // function takes into account velocity limitations due to traffic lights.
  __device__ uint32_t max_velocity() const {
    return controller_max_velocity_ < max_velocity_
        ? controller_max_velocity_
        : max_velocity_;
  }

  // Sets the maximum temporary speed limit (traffic controller).
  __device__ void set_controller_max_velocity(uint32_t velocity) {
    controller_max_velocity_ = velocity;
  }

  // Removes the maximum temporary speed limit.
  __device__ void remove_controller_max_velocity() {
    controller_max_velocity_ = max_velocity_;
  }

  // Incoming cells.
  array_(Cell*, kArrayInlineSizeIncomingCells, inline_soa) incoming_cells_;
  uint32_t_ num_incoming_cells_;
  __device__ uint32_t num_incoming_cells() const {
    return num_incoming_cells_;
  }
  __device__ Cell* incoming_cell(uint32_t index) const {
    return incoming_cells_[index];
  }

  // Outgoing cells.
  array_(Cell*, kArrayInlineSizeOutgoingCells, inline_soa) outgoing_cells_;
  uint32_t_ num_outgoing_cells_;
  __device__ uint32_t num_outgoing_cells() const {
    return num_outgoing_cells_;
  }
  __device__ Cell* outgoing_cell(uint32_t index) const {
    return outgoing_cells_[index];
  }

  // The car that is currently occupying this cell (if any).
  field_(Car*) car_;

  // A car enters this cell.
  __device__ void occupy(Car* car) {
    car_ = car;
    is_free_ = false;
  }

  // A car leaves this cell.
  __device__ void release() {
    car_ = nullptr;
    is_free_ = true;
  }

  // The type of this cell according to OSM data.
  field_(Type) type_;
  __device__ Type type() const { return type_; }

  // x and y coordinates, only for rendering and debugging purposes.
  double_ x_;
  double_ y_;
};

IKRA_DEVICE_STORAGE(Cell);

class Car : public SoaLayout<
    Car, kNumCars, kAddressModeZero,
    StaticStorageWithArena<kNumCars*50*sizeof(uint32_t)>> {
 public:
  IKRA_INITIALIZE_CLASS

  __device__ __host__ Car(bool is_active, uint32_t velocity,
                          uint32_t max_velocity, uint32_t random_state,
                          Cell* position)
      : is_active_(is_active), velocity_(velocity), path_length_(0),
        path_(max_velocity), random_state_(random_state), position_(position),
        max_velocity_(max_velocity) {}

  // If a car enters a sink, it is removed from the simulation (inactive)
  // for a short time.
  bool_ is_active_;
  __device__ bool is_active() const { return is_active_; }

  // The velocity of the car in cells/iteration.
  uint32_t_ velocity_;
  __device__ uint32_t velocity() const { return velocity_; }

  // The max. possible velocity of this car.
  uint32_t_ max_velocity_;
  __device__ uint32_t max_velocity() const { return max_velocity_; }

  // An array of cells that the car will move onto next.
  array_(Cell*, kArrayInlineSizePath, inline_soa) path_;
  uint32_t_ path_length_;

  // The current position of the car.
  field_(Cell*) position_;
  __device__ Cell* position() const { return position_; }

  // Every car has a random state to allow for reproducible results.
  uint32_t_ random_state_;

  __device__ uint32_t rand32() {
    // Advance and return random state.
    // Source: https://en.wikipedia.org/wiki/Lehmer_random_number_generator
    random_state_ = static_cast<uint32_t>(
        static_cast<uint64_t>(random_state()) * 279470273u) % 0xfffffffb;
    return random_state_;
  }

  __device__ uint32_t rand32(uint32_t max_value) {
    return rand32() % max_value;
  }

  __device__ uint32_t random_state() const {
    return random_state_;
  }

  __device__ void step_prepare_path() {
    step_initialize_iteration();
    step_accelerate();
    step_extend_path();
    step_constraint_velocity();
  }

  __device__ Cell* next_step(Cell* cell);

  __device__ void step_initialize_iteration();

  __device__ void step_accelerate();

  __device__ void step_extend_path();

  __device__ void step_constraint_velocity();

  __device__ void step_move();

  __device__ void step_reactivate();
};

IKRA_DEVICE_STORAGE(Car);


__device__ Cell* Car::next_step(Cell* position) {
  // Random walk.
  uint32_t num_cells = position->num_outgoing_cells();
  return position->outgoing_cell(rand32(num_cells));
}

__device__ void Car::step_initialize_iteration() {
  // Reset calculated path. This forces cars with a random moving behavior to
  // select a new path in every iteration. Otherwise, cars might get "stucjk"
  // on a full network if many cars are waiting for the one in front of them in
  // a cycle.
  // TODO: Check if we can keep the path at least partially somehow.
  path_length_ = 0;
}

__device__ void Car::step_accelerate() {
  // Speed up the car by 1 or 2 units.
  uint32_t speedup = rand32(2) + 1;
  velocity_ = max_velocity_ < velocity_ + speedup
      ? static_cast<uint32_t>(max_velocity_) : velocity_ + speedup;
}

__device__ void Car::step_extend_path() {
  Cell* cell = position_;

  for (uint32_t i = 0; i < velocity_; ++i) {
    if (cell->is_sink()) {
      break;
    }

    cell = next_step(cell);
    path_[i] = cell;
    path_length_ = path_length_ + 1;
  }

  velocity_ = path_length_;
}

__device__ void Car::step_constraint_velocity() {
  // This is actually only needed for the very first iteration, because a car
  // may be positioned on a traffic light cell.
  // TODO: Why does the implicit type cast not work here?
  if (velocity_ > position()->max_velocity()) {
    velocity_ = position()->max_velocity();
  }

  uint32_t distance = 0;
  while (distance < velocity_) {
    // Invariant: Movement of up to `distance` many cells at `velocity_`
    //            is allowed.
    // Now check if next cell can be entered.
    Cell* next_cell = path_[distance];

    // Avoid collision.
    if (!next_cell->is_free()) {
      // Cannot enter cell.
      velocity_ = distance;
      --distance;
      break;
    } // else: Can enter next cell.

    if (velocity_ > next_cell->max_velocity()) {
      // Car is too fast for this cell.
      if (next_cell->max_velocity() > distance) {
        // Even if we slow down, we would still make progress.
        velocity_ = next_cell->max_velocity();
      } else {
        // Do not enter the next cell.
        velocity_ = distance;
        --distance;
        break;
      }
    }

    ++distance;
  }

  --distance;
  assert(distance < velocity_);
}

__device__ void Car::step_move() {
  Cell* cell;
  for (int i = 0; i < velocity_; ++i) {
    // TODO: Add check here to see if cell is free.
    cell = path_[i];
  }

  position()->release();
  cell->occupy(this);
  position_ = cell;

  if (position()->is_sink()) {
    // Remove car from the simulation. Will be added again in the next
    // iteration.
    position()->release();
    path_length_ = 0;
    is_active_ = false;
  }
}

__device__ void Car::step_reactivate() {
  // TODO
}

#include "simulation_adapter.cuh"
#include "traffic-simulation/traffic_aos_int_cuda.h"

using IndexType = unsigned int;

// Data storage.
namespace simulation {
namespace aos_int {
extern IndexType s_size_Cell;
extern IndexType s_size_outgoing_cells;
extern IndexType s_size_incoming_cells;
extern IndexType s_size_Car;
extern IndexType s_size_car_paths;
extern IndexType s_size_inactive_cars;
extern IndexType s_size_TrafficLight;
extern IndexType s_size_PriorityYieldTrafficController;
extern IndexType s_size_SharedSignalGroup;
extern IndexType s_size_traffic_light_signal_groups;
extern IndexType s_size_priority_ctrl_signal_groups;
extern IndexType s_size_signal_group_cells;
}  // namespace aos_int

namespace aos_int_cuda {
extern Cell* dev_Cell;
extern IndexType* dev_outgoing_cells;
extern IndexType* dev_incoming_cells;
extern Car* dev_Car;
extern IndexType* dev_car_paths;
extern IndexType* dev_inactive_cars;
extern TrafficLight* dev_TrafficLight;
extern PriorityYieldTrafficController* dev_PriorityYieldTrafficController;
extern SharedSignalGroup* dev_SharedSignalGroup;
extern IndexType* dev_traffic_light_signal_groups;
extern IndexType* dev_priority_ctrl_signal_groups;
extern IndexType* dev_signal_group_cells;
}  // namespace aos_int_cuda
}  // namespace simulation

/*
  __host__ __device__ Cell(uint32_t max_velocity, double x, double y,
                           uint32_t num_incoming, Cell** incoming,
                           uint32_t num_outgoing, Cell** outgoing,
                           Car* car, bool is_free, bool is_sink,
                           Type type = kResidential)
*/

__global__ void convert_to_ikra_cpp_cells(
    IndexType s_size_Cell,
    simulation::aos_int_cuda::Cell* s_Cell,
    IndexType s_size_outgoing_cells,
    IndexType* s_outgoing_cells,
    IndexType s_size_incoming_cells,
    IndexType* s_incoming_cells) {
  unsigned int tid = blockIdx.x *blockDim.x + threadIdx.x;

  if (tid < s_size_Cell) {
    simulation::aos_int_cuda::Cell& cell = s_Cell[tid];
    Car* car_ptr = cell.car_ == 4294967295
      ? nullptr : Car::get_uninitialized(cell.car_);

    Cell* new_cell = new(Cell::get_uninitialized(tid)) Cell(
        cell.max_velocity_, cell.x_, cell.y_,
        cell.num_incoming_cells_,
        s_incoming_cells + cell.first_incoming_cell_idx_,
        cell.num_outgoing_cells_,
        s_outgoing_cells + cell.first_outgoing_cell_idx_,
        car_ptr, cell.is_free_, cell.is_sink_,
        (Cell::Type) cell.type_);
    assert(new_cell->id() == tid);
  }

  if (tid == 0) {
    Cell::storage().increase_size(s_size_Cell);
  }
}

/*
  __device__ __host__ Car(bool is_active, uint32_t velocity,
                          uint32_t max_velocity, uint32_t random_state,
                          Cell* position)
                          */

__global__ void convert_to_ikra_cpp_cars(
    IndexType s_size_Car,
    simulation::aos_int_cuda::Car* s_Car) {
  unsigned int tid = blockIdx.x *blockDim.x + threadIdx.x;

  if (tid < s_size_Car) {
    simulation::aos_int_cuda::Car& car = s_Car[tid];
    Cell* cell_ptr = car.position_ == 4294967295
      ? nullptr : Cell::get_uninitialized(car.position_);

    Car* new_car = new(Car::get_uninitialized(tid)) Car(
        car.is_active_, car.velocity_, car.max_velocity_,
        car.random_state_, cell_ptr);
    assert(new_car->id() == tid);
  }

  if (tid == 0) {
    Car::storage().increase_size(s_size_Car);
  }
}

int main(int argc, char** argv) {
  Cell::initialize_storage();
  Car::initialize_storage();

  load_simulation(argc, argv);

  assert(simulation::aos_int::s_size_Car <= kNumCars);
  assert(simulation::aos_int::s_size_Cell <= kNumCells);

  convert_to_ikra_cpp_cells<<<kNumCells/1024 + 1, 1024>>>(
      simulation::aos_int::s_size_Cell,
      simulation::aos_int_cuda::dev_Cell,
      simulation::aos_int::s_size_outgoing_cells,
      simulation::aos_int_cuda::dev_outgoing_cells,
      simulation::aos_int::s_size_incoming_cells,
      simulation::aos_int_cuda::dev_incoming_cells);
  gpuErrchk(hipDeviceSynchronize());

  convert_to_ikra_cpp_cars<<<kNumCars/1024 + 1, 1024>>>(
      simulation::aos_int::s_size_Car,
      simulation::aos_int_cuda::dev_Car);
  gpuErrchk(hipDeviceSynchronize());

  // Now start simulation.
  cuda_execute(&Car::step_prepare_path);
  gpuErrchk(hipDeviceSynchronize());
}
