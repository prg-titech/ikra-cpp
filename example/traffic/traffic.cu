#include "hip/hip_runtime.h"
#include <limits>
#include <iostream>

#include "executor/executor.h"
#include "soa/soa.h"
#include "executor/cuda_executor.h"

#include "benchmark.h"
#include "configuration.h"
#include "simulation_adapter.h"
#include "traffic-simulation/traffic_aos_int_cuda.h"

using ikra::soa::SoaLayout;
using ikra::soa::kAddressModeZero;
using ikra::soa::StaticStorageWithArena;

class Car;

// TODO: Figure out how extern __device__ variables work in CUDA, so that we
// can use normal header files instead of textual headers.
#include "entities/cell.inc"
#include "entities/car.inc"
#include "entities/shared_signal_group.inc"
#include "entities/traffic_light.inc"
#include "entities/priority_yield_traffic_controller.inc"
#include "entities/simulation.inc"
#include "simulation_converter.inc"
#include "statistics.inc"


void run_traffic_controllers() {
  cuda_execute(&Simulation::step_random_state);
  cuda_execute(&TrafficLight::step);
  cuda_execute(&PriorityYieldTrafficController::step);
  hipDeviceSynchronize();
}

void run_cars() {
  cuda_execute(&Car::step_prepare_path);
  hipDeviceSynchronize();

  cuda_execute(&Car::step_move);
  hipDeviceSynchronize();

  cuda_execute(&Car::step_reactivate);
  hipDeviceSynchronize();
}

void benchmark() {
  uint64_t time_controllers[kNumBenchmarkRuns] = {0};
  uint64_t time_cars[kNumBenchmarkRuns] = {0};
  uint64_t time_total[kNumBenchmarkRuns] = {0};

  for (uint32_t r = 0; r < kNumBenchmarkRuns; ++r) {
    Cell::initialize_storage();
    Car::initialize_storage();
    SharedSignalGroup::initialize_storage();
    TrafficLight::initialize_storage();
    PriorityYieldTrafficController::initialize_storage();

    convert_simulation();
    cuda_execute(&TrafficLight::initialize);
    cuda_execute(&PriorityYieldTrafficController::initialize);
    hipDeviceSynchronize();

    for (uint32_t i = 0; i < kNumIterations; ++i) {
      uint64_t t_ctrl = measure<>::execution(run_traffic_controllers);
      time_controllers[r] += t_ctrl;

      uint64_t t_car = measure<>::execution(run_cars);
      time_cars[r] += t_car;

      time_total[r] = time_controllers[r] + time_cars[r];
    }
    gpuErrchk(hipPeekAtLastError());
  }

  // Find best run.
  uint64_t best_time = std::numeric_limits<uint64_t>::max();
  uint32_t best_index = -1;
  for (uint32_t r = 0; r < kNumBenchmarkRuns; ++r) {
    if (time_total[r] < best_time) {
      best_time = time_total[r];
      best_index = r;
    }
  }

  // Print best run.
  print_statistics(time_cars[best_index]/1000,
                   time_controllers[best_index]/1000,
                   time_total[best_index]/1000);
}

int main(int argc, char** argv) {
  load_simulation(argc, argv, kNumCars);
  benchmark();

  print_histograms_1<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());
  print_histograms_2<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());
  print_histograms_3<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());
  print_histograms_4<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());
  print_histograms_5<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());
  caluclate_checksum<<<1,1>>>();
  gpuErrchk(hipDeviceSynchronize());
}
