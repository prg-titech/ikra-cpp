#include "hip/hip_runtime.h"
#include <limits>
#include <math.h>
#include <stdint.h>

#include "bfs_loader.h"
#include "executor/cuda_executor.h"
#include "executor/executor.h"
#include "soa/soa.h"

static const int kMaxDegree = 10;
static const int kMaxVertices = 20000;

using ikra::soa::IndexType;
using ikra::soa::SoaLayout;
using ikra::executor::execute;

class Vertex : public SoaLayout<Vertex, kMaxVertices> {
 public:
  IKRA_INITIALIZE_CLASS

  Vertex(const std::vector<IndexType>& neighbors) {
    // If this check fails, we the dataset cannot be run with this
    // implementation.
    assert(neighbors.size() <= kMaxDegree);
    adj_list_size_ = neighbors.size();

    for (int i = 0; i < num_neighbors(); ++i) {
      Vertex* vertex = Vertex::get_uninitialized(neighbors[i]);
      adj_list_[i] = vertex;
    }
  }

  __host__ __device__ int num_neighbors() {
    return adj_list_size_;
  }

  // Visit the vertex, i.e., update the distances of all neighbors if this
  // vertex is in the frontier, as indicated by the "iteration" field. Returns
  // "true" if at least one neighbor was updated.
  __device__ bool visit(int iteration) {
    bool updated = false;

    if (distance_ == iteration) {
      for (int i = 0; i < num_neighbors(); ++i) {
        Vertex* neighbor = adj_list_[i];
        updated |= neighbor->update_distance(distance_ + 1);
      }
    }

    return updated;
  }

  void print_distance() {
    printf("distance[%lu] = %i\n", id(), (int) distance_);
  }

  void set_distance(int value) {
    distance_ = value;
  }

  __device__ bool update_distance(int distance) {
    if (distance < distance_) {
      distance_ = distance;
      return true;
    } else {
      return false;
    }
  }

  int_ distance_ = std::numeric_limits<int>::max();
  int_ adj_list_size_;

  // By default a SOA array.
  array_(Vertex*, kMaxDegree) adj_list_;
};

IKRA_DEVICE_STORAGE(Vertex)


int run() {
  int iteration = 0;
  bool running = true;

  while (running) {
    auto reducer = [](bool a, bool b) { return a || b; };
    running = cuda_execute_and_reduce(&Vertex::visit,
                                      reducer,
                                      iteration);

    ++iteration;
  }

  return iteration;
}

int main(int argc, char* argv[]) {
  // Load vertices from file.
  if (argc != 4) {
    printf("Usage: %s filename num_vertices start_vertex\n", argv[0]);
    exit(1);
  }

  Vertex::initialize_storage();
  load_file<Vertex>(argv[1], atoi(argv[2]));

  // Set start vertex.
  Vertex* start_vertex = Vertex::get(atoi(argv[3]));
  start_vertex->set_distance(0);

  // Start algorithm.
  int iterations = run();

  // Note: execute is host side, cuda_execute is device side.
  printf("Iterations: %i\n", iterations);
  execute(&Vertex::print_distance);

  // Ensure nothing went wrong on the GPU.
  gpuErrchk(hipPeekAtLastError());
}
